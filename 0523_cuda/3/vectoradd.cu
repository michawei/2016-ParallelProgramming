
#include <hip/hip_runtime.h>
#include <stdio.h>

/* experiment with N */
/* how large can it be? */
//#define N (2048*2048)
#define N 10240
#define THREADS_PER_BLOCK 4

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d\n", index);
	if ( index < N ){
		for ( int i = 0 ; i < index ; i++ )
			a[index] += i;

		//__syncthreads();
		//c[index] = a[N-1-index];
	}
}

__global__ void add_2(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[N-1-index];
}


int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ ) {
		a[i] = -1;
		b[i] = -1;
		c[i] = -1;
	}

	hipStream_t streams[2];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);
	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );
	printf("%d %d\n", (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */
	add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK , 0, streams[0]>>>( d_a, d_b, d_c );

	hipStreamSynchronize(streams[0]);
	//cudaMemcpy( a, d_a, size, cudaMemcpyDeviceToHost );

	add_2<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK , 0, streams[1]>>>( d_a, d_b, d_c );

	hipStreamSynchronize(streams[1]);
	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	int sum = 0;
	for ( int i = 0 ; i < N ; i++ ){
		sum += c[i];
	}
	printf( "sum = %d\n", sum);

	// printf( "c[0] = %d\n", 0, c[0]);
	// printf( "c[%d] = %d\n", N-1,c[N-1]);

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
