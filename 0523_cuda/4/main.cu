#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include "utils.h"
 
#define MAXN 10000005
#define BLOCK 500
uint32_t prefix_sum[MAXN];

void output(uint32_t presum[], int n) {
    uint32_t hash = 0;
    for (int i = 1; i <= n; i++)
        hash += presum[i] * i;
    printf("%" PRIu32 "\n", hash);
}

__global__ void PrifixSum_1(uint32_t *prefixSum, uint32_t *preBlockSum, int n) {

    int start = blockIdx.x * BLOCK;
    //printf("sss = %d\n", start);
    int end = start + BLOCK;
    end = (end < n) ? end : n;

    uint32_t sum = 0;
    for ( int i = start ; i < end ; i++ ) {
        sum += prefixSum[i];
        prefixSum[i] = sum;
    }
    preBlockSum[blockIdx.x] = sum;
}

__global__ void PrifixSum_2(uint32_t *prefixSum, uint32_t *preBlockSum, int n) {

    int start = blockIdx.x * BLOCK;
    //printf("sss = %d\n", start);
    int end = start + BLOCK;
    end = (end < n) ? end : n;
    uint32_t add = (blockIdx.x == 0) ? 0 : preBlockSum[blockIdx.x - 1];
    for ( int i = start ; i < end ; i++ ) {
        prefixSum[i] += add;
    }
}

int main() {
    int n;
    uint32_t key;
    while (scanf("%d %" PRIu32, &n, &key) == 2) {

        //printf("---------------------------------\n");

        for ( int i = 0 ; i <= n ; i++ ) {
            prefix_sum[i] = encrypt(i, key);
            //printf("%u ", prefix_sum[i]);
        }
        n += 1;
        //printf("\n");
        // upper-bound int
        int block = (n + (BLOCK - 1)) / BLOCK;
        //printf("%d %d\n", n, block);
        uint32_t pre_block_sum[block];

        hipSetDevice(2);
        uint32_t *d_prefix_sum;
        uint32_t *d_pre_block_sum;
        int size = n * sizeof(uint32_t);
        int second_size = block * sizeof(uint32_t);

        hipMalloc((void **) &d_prefix_sum, size);
        hipMalloc((void **) &d_pre_block_sum, second_size);

        hipMemcpy(d_prefix_sum, prefix_sum, size, hipMemcpyHostToDevice);
        PrifixSum_1<<< block, BLOCK >>>(d_prefix_sum, d_pre_block_sum, n);
        //hipMemcpy(prefix_sum, d_prefix_sum, size, hipMemcpyDeviceToHost);
        hipMemcpy(pre_block_sum, d_pre_block_sum, second_size, hipMemcpyDeviceToHost);

        uint32_t sum = 0;
        for ( int i = 0 ; i < block ; i++ ) {
            sum += pre_block_sum[i];
            pre_block_sum[i] = sum;
        }

        // for ( int i = 0 ; i < n ; i++ ) {
        //     printf("%u ", prefix_sum[i]);
        // }
        // printf("\n");

        // for ( int i = 0 ; i < block ; i++ ) {
        //     printf("%u ", pre_block_sum[i]);
        // }
        // printf("\n***\n");

        hipMemcpy(d_pre_block_sum, pre_block_sum, second_size, hipMemcpyHostToDevice);
        PrifixSum_2<<< block, 1 >>>(d_prefix_sum, d_pre_block_sum, n);
        hipMemcpy(prefix_sum, d_prefix_sum, size, hipMemcpyDeviceToHost);

        // for ( int i = 0 ; i < block ; i++ ) {
        //     printf("%u ", pre_block_sum[i]);
        // }
        // printf("\n***\n");

        // for ( int i = 0 ; i < n ; i++ ) {
        //     printf("%u ", prefix_sum[i]);
        // }
        // printf("\n");
        output(prefix_sum, n);
        hipFree(d_prefix_sum);
        hipFree(d_pre_block_sum);
    }
    return 0;
}
