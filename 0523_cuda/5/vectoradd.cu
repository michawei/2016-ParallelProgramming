
#include <hip/hip_runtime.h>
#include <stdio.h>

/* experiment with N */
/* how large can it be? */
#define N 1024
#define THREADS_PER_BLOCK 1024

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int tmpA[N];
	__shared__ int tmpB[N];

	tmpA[index] = a[index];
	tmpB[index] = b[index];

	__syncthreads();

	for ( int i = 0 ; i < N ; i++ )
		tmpB[index] += tmpA[i];

	//__syncthreads();

	b[index] = tmpB[index];

	__syncthreads();

	// for ( int i = 0 ; i < N ; i++ )
	// 	b[index] += a[i];
}

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = i;
		b[i] = 0;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	for ( int i=0 ; i < 10*N ; i++ )
		add<<< 1, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( b, d_b, size, hipMemcpyDeviceToHost );


	printf( "b[0] = %d\n",0,b[0] );
	printf( "b[%d] = %d\n",N-2, b[N-2] );
	printf( "b[%d] = %d\n",N-1, b[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
