#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
	/* Get Device Num */
	int cudaDeviceNum = 0;
	hipGetDeviceCount(&cudaDeviceNum);
	printf("%d devices found supporting CUDA\n", cudaDeviceNum);

	if ( cudaDeviceNum == 0 ) {
		printf("No GPU\n");
		return 0;
	}

	for (int i = 0; i < cudaDeviceNum; i++) {
		hipDeviceProp_t deviceInfo;
		hipGetDeviceProperties(&deviceInfo, i);
		printf("----------------------------------\n");
		printf("Device %s\n", deviceInfo.name);
		printf("----------------------------------\n");
		printf(" Device memory: \t%zu\n", deviceInfo.totalGlobalMem);
		printf(" Memory per-block: \t%zu\n", deviceInfo.sharedMemPerBlock);
		printf(" Register per-block: \t%d\n", deviceInfo.regsPerBlock);
		printf(" Warp size: \t\t%d\n", deviceInfo.warpSize);
		printf(" Memory pitch: \t\t%zu\n", deviceInfo.memPitch);
		printf(" Constant Memory: \t%zu\n", deviceInfo.totalConstMem);
		printf(" Max thread per-block: \t%d\n", deviceInfo.maxThreadsPerBlock);
		printf(" Max thread dim: \t%d / %d / %d\n", deviceInfo.maxThreadsDim[0], deviceInfo.maxThreadsDim[1], deviceInfo.maxThreadsDim[2]);
		printf(" Max grid size: \t%d / %d / %d\n", deviceInfo.maxGridSize[0], deviceInfo.maxGridSize[1], deviceInfo.maxGridSize[2]);
		printf(" Ver: \t\t\t%d.%d\n", deviceInfo.major, deviceInfo.minor);
		printf(" Clock: \t\t%d\n", deviceInfo.clockRate);
		printf(" Texture Alignment: \t%zu\n", deviceInfo.textureAlignment);
	}

	return 0;
}