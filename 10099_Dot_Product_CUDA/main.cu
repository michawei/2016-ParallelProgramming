#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>
#include "utils.h"
 
#define MAXN 16777216
#define THREADS_PER_BLOCK 512

__global__ void dot(int *A, int *B, int *C) {
	/* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	// int index = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int tmp[50000];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	C[idx] = A[idx] * B[idx];
}

int A[MAXN], B[MAXN], C[MAXN];

int main(int argc, char *argv[]) {

	hipSetDevice(2);
	int *d_A, *d_B, *d_C;
	int size = MAXN * sizeof(int);
	/* allocate space for device copies o f a, b, c */
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);

	int N;
	uint32_t key1, key2;

	while (scanf("%d %" PRIu32 " %" PRIu32, &N, &key1, &key2) == 3) {
		//int chunk = N / 4;
		for (int i = 0; i < N; i++) {
			A[i] = encrypt(i, key1);
			B[i] = encrypt(i, key2);
		}

		int local_size = N * sizeof(int);

		hipMemcpy(d_A, A, local_size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, B, local_size, hipMemcpyHostToDevice);
 
		dot<<< (N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_A, d_B, d_C);
		//hipDeviceSynchronize();
		//dot<<< N, 1>>>(d_A, d_B, d_C);
		//hipDeviceSynchronize();
		hipMemcpy(C, d_C, local_size, hipMemcpyDeviceToHost);

		uint32_t sum = 0;
		for ( int i = 0; i < N; i++ )
			sum += C[i];
		printf("%" PRIu32 "\n", sum);
	}

	// free(A);
	// free(B);
	// free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}